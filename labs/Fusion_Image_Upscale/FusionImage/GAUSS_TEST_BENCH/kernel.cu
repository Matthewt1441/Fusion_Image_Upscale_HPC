
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>

#include <fstream>
#include <string>

typedef struct __align__(4) { // Or alignas(4) in C++11
    unsigned char r;
    unsigned char g;
    unsigned char b;
    unsigned char a; // Padding to ensure 4-byte alignment
}RGBA_t;

char* readPPM(char* filename, int* width, int* height) {
    //std::ifstream file(filename, std::ios::binary);

    std::ifstream file(filename, std::ios::binary); // open the file and throw exception if it doesn't exist
    if (file.fail())
        throw "File failed to open";

    std::string magicNumber;
    int maxColorValue;
    int w = 0;
    int h = 0;

    file >> magicNumber;
    file >> w >> h >> maxColorValue;

    file.get(); // skip the trailing white space

    size_t size = w * h * 3;
    char* pixel_data = new char[size];

    file.read(pixel_data, size);

    *width = w;
    *height = h;

    return pixel_data;
}

char* readPPMGray(char* filename, int* width, int* height) {
    //std::ifstream file(filename, std::ios::binary);

    std::ifstream file(filename, std::ios::binary); // open the file and throw exception if it doesn't exist
    if (file.fail())
        throw "File failed to open";

    std::string magicNumber;
    int maxColorValue;
    int w = 0;
    int h = 0;

    file >> magicNumber;
    file >> w >> h >> maxColorValue;

    file.get(); // skip the trailing white space

    size_t size = w * h;
    char* pixel_data = new char[size];

    file.read(pixel_data, size);

    *width = w;
    *height = h;

    return pixel_data;
}

void writePPM(char* filename, char* img_data, int width, int height)
{
    std::ofstream file(filename, std::ios::binary);
    if (file.fail())
        throw "File failed to open";

    file << "P6" << "\n" << width << " " << height << "\n" << 255 << "\n";

    size_t size = (width) * (height) * 3;

    file.write(img_data, size);
}

void writePPMGrey(char* filename, char* img_data, int width, int height)
{
    std::ofstream file(filename, std::ios::binary);
    if (file.fail())
        throw "File failed to open";

    file << "P5" << "\n" << width << " " << height << "\n" << 255 << "\n";

    size_t size = (width) * (height);

    file.write(img_data, size);
}

__global__ void RGB2GreyscaleKernel(unsigned char* rgb_img, unsigned char* grey_img, int width, int height)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < height && Col < width)
    {
        int rgbidx = rgbidx = 3 * (Row * width + Col);
        grey_img[Row * width + Col] = (21 * rgb_img[rgbidx + 0] /100) + (71 * rgb_img[rgbidx + 1] / 100) + (7 * rgb_img[rgbidx + 2] / 100);
    }
}

//Initial Naive approach
__global__ void rgbToRGBA_Kernel(RGBA_t* d_RGBA_img, unsigned char* d_rgb_img, int numpixels)
{
    // Each thread processes one pixel
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    int sharedIdx = tid * 3;

    // Shared memory for the current block of RGB values
    extern __shared__ unsigned char sharedRGB_char[];

    // Load data into shared memory
    if(idx < numpixels)
    {
        sharedRGB_char[sharedIdx + 0] = d_rgb_img[idx * 3 + 0];
        sharedRGB_char[sharedIdx + 1] = d_rgb_img[idx * 3 + 1];
        sharedRGB_char[sharedIdx + 2] = d_rgb_img[idx * 3 + 2];
    }

    // Synchronize to ensure all threads have loaded their data into shared memory
    __syncthreads();

    // Now process the RGB to RGBA conversion in shared memory
    if(idx < numpixels) {
        // Read RGB values from shared memory
        unsigned char r = sharedRGB_char[sharedIdx + 0];
        unsigned char g = sharedRGB_char[sharedIdx + 1];
        unsigned char b = sharedRGB_char[sharedIdx + 2];

        // Write to RGBA array (global memory)
        d_RGBA_img[idx].r = r;
        d_RGBA_img[idx].g = g;
        d_RGBA_img[idx].b = b;
        d_RGBA_img[idx].a = 255;  // Alpha is fully opaque
    }
}


//Initial Naive approach
__global__ void rgbaToRGB_Kernel(unsigned char* d_rgb_img, RGBA_t* d_rgba_img, int numpixels)
{
    // Each thread processes one pixel
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int sharedIdx = threadIdx.x;

    // Shared memory for the current block of RGBA values
    __shared__ RGBA_t sharedRGB[256];  // Assuming block size is 256 threads

    // Load data into shared memory
    if(idx < numpixels)
    {
        sharedRGB[sharedIdx] = d_rgba_img[idx];
    }

    // Synchronize to ensure all threads have loaded their data into shared memory
    __syncthreads();

    // Now process the RGBA to RGB conversion in shared memory
    if(idx < numpixels) {
        // Read RGBA values from shared memory
        RGBA_t rgba_val = sharedRGB[sharedIdx];

        // Write to RGB array (global memory)
        d_rgb_img[idx * 3 + 0] = rgba_val.r;
        d_rgb_img[idx * 3 + 1] = rgba_val.g;
        d_rgb_img[idx * 3 + 2] = rgba_val.b;
    }
}
void Image_Compare(unsigned char* img1, unsigned char* img2, int width, int height)
{
    int idx = 0;
    int y;
    int x;
    bool pass = true;
    for( y = 0; y < height; y++) 
    {
        for(x = 0; x < width; x++)
        {
            idx = y * width + x;
            char img1_r = img1[idx + 0];
            char img1_g = img1[idx + 1];
            char img1_b = img1[idx + 2];
            char img2_r = img2[idx + 0];
            char img2_g = img2[idx + 1];
            char img2_b = img2[idx + 2];

            if((img2_r < img1_r - 5) || (img2_r > img1_r + 5))
            {
                pass = false;
                goto LOOP_EXIT;
            }

            if((img2_g < img1_g - 5) || (img2_g > img1_g + 5))
            {
                pass = false;
                goto LOOP_EXIT;
            }

            if((img2_b < img1_b - 5) || (img2_b > img1_b + 5))
            {
                pass = false;
                goto LOOP_EXIT;
            }

        }
    }

LOOP_EXIT:
    if(!pass)
    {
        printf("Images do not match at pixel X: %d, Y: %d, Img1 [%d, %d, %d], Img2 [%d, %d, %d]\n", x, y, img1[idx + 0], img1[idx + 1], img1[idx + 2], img2[idx + 0], img2[idx + 1], img2[idx + 2]);

    }
    else
    {
        printf("Images match!\n");
    }

}

void Grey_Image_Compare(unsigned char* img1, unsigned char* img2, int width, int height)
{
    int idx = 0;
    int y;
    int x;
    bool pass = true;
    for( y = 0; y < height; y++) 
    {
        for(x = 0; x < width; x++)
        {
            idx = y * width + x;

            if(img1[idx] < img2[idx] - 5 || img1[idx] > img2[idx] + 5 )
            {
                pass = false;
                goto GREY_LOOP_EXIT;
            }


        }
    }

GREY_LOOP_EXIT:
    if(!pass)
    {
        printf("Images do not match at pixel X: %d, Y: %d, Img1 [%d, %d, %d], Img2 [%d, %d, %d]\n", x, y, img1[idx + 0], img1[idx + 1], img1[idx + 2], img2[idx + 0], img2[idx + 1], img2[idx + 2]);

    }
    else
    {
        printf("Images match!\n");
    }
}

__device__ float cubicInterpolateDevice(float p[4], float x)
{
    float output = p[1] + 0.5 * x * (p[2] - p[0] + x * (2.0 * p[0] - 5.0 * p[1] + 4.0 * p[2] - p[3] + x * (3.0 * (p[1] - p[2]) + p[3] - p[0])));

    output = output * ((output <= 255.0) && (output >= 0.0)) + 255 * (output > 255.0) + 0 * (output < 0);
    return output;
}

__device__ float bicubicInterpolateDevice(float p[4][4], float y, float x)
{
    float arr[4];
    arr[0] = cubicInterpolateDevice(p[0], x);
    arr[1] = cubicInterpolateDevice(p[1], x);
    arr[2] = cubicInterpolateDevice(p[2], x);
    arr[3] = cubicInterpolateDevice(p[3], x);
    return cubicInterpolateDevice(arr, y);
}

__global__ void bicubicInterpolationKernel(unsigned char* big_img_data, unsigned char* img_data, int big_width, int big_height, int width, int height, int scale)
{

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    int input_x = 0;
    int input_y = 0;

    int window_x = 0;
    int window_y = 0;
    
    int output_x = Col;
    int output_y = Row;


    float window_r[4][4];
    float window_g[4][4];
    float window_b[4][4];

    int sample_x = 0;
    int sample_y = 0;

    for (window_y = 0; window_y < 4; window_y++)
    {
        for (window_x = 0; window_x < 4; window_x++)
        {
            window_r[window_y][window_x] = 0;
            window_g[window_y][window_x] = 0;
            window_b[window_y][window_x] = 0;
        }
    }

    if(output_y < big_height &&  output_x < big_width)
    {
        //Calculate starting index for windows
        float interpolated_x = (float)(output_x / (scale * 1.0));
        float interpolated_y = (float)(output_y / (scale * 1.0));

        int input_block_start_idx_x = (output_x / scale);
        int input_block_start_idx_y = (output_y / scale);

        float dx = interpolated_x - input_block_start_idx_x;
        float dy = interpolated_y - input_block_start_idx_y;

        //We are within a block of the input image, therefore fill windows
        for(window_y = -1; window_y < 3; window_y++)
        {
            for(window_x = -1; window_x < 3; window_x++)
            {
                //Calculate Input Image index
                input_x = input_block_start_idx_x + window_x;
                input_y = input_block_start_idx_y + window_y;

                // Fill window with Nearest Neighbor edge behavior
                if(input_x < 0 || input_x >= width)
                {
                    // Find nearest in-bounds pixel
                    input_x = (input_x < 0) ? 0 : width - 1;
                }
                // Fill window with Nearest Neighbor edge behavior
                if(input_y < 0 || input_y >= height)
                {
                    // Find nearest in-bounds pixel
                    input_y = (input_y < 0) ? 0 : height - 1;
                }

                window_r[window_y + 1][window_x + 1] = (float)img_data[3 * (input_y * width + input_x) + 0];    //R
                window_g[window_y + 1][window_x + 1] = (float)img_data[3 * (input_y * width + input_x) + 1];    //G
                window_b[window_y + 1][window_x + 1] = (float)img_data[3 * (input_y * width + input_x) + 2];    //B
            }
        }

        float r = bicubicInterpolateDevice(window_r, dy, dx);
        float g = bicubicInterpolateDevice(window_g, dy, dx);
        float b = bicubicInterpolateDevice(window_b, dy, dx);

        big_img_data[3 * (output_y * big_width + output_x) + 0] = (unsigned char)r;
        big_img_data[3 * (output_y * big_width + output_x) + 1] = (unsigned char)g;
        big_img_data[3 * (output_y * big_width + output_x) + 2] = (unsigned char)b;
    }
}

__device__ float cubicInterpolateDevice_GreyCon(float p[4], float x)
{
    float output = p[1] + 0.5 * x * (p[2] - p[0] + x * (2.0 * p[0] - 5.0 * p[1] + 4.0 * p[2] - p[3] + x * (3.0 * (p[1] - p[2]) + p[3] - p[0])));

    output = output * ((output <= 255.0) && (output >= 0.0)) + 255 * (output > 255.0) + 0 * (output < 0);
    return output;
}

__device__ float bicubicInterpolateDevice_GreyCon(float p[4][4], float y, float x)
{
    float arr[4];
    arr[0] = cubicInterpolateDevice_GreyCon(p[0], x);
    arr[1] = cubicInterpolateDevice_GreyCon(p[1], x);
    arr[2] = cubicInterpolateDevice_GreyCon(p[2], x);
    arr[3] = cubicInterpolateDevice_GreyCon(p[3], x);
    return cubicInterpolateDevice_GreyCon(arr, y);
}

__global__ void bicubicInterpolation_GreyCon_Kernel_RGBA(RGBA_t* big_img_data, unsigned char* grey_big_img_data, RGBA_t* img_data, int big_width, int big_height, int width, int height, int scale)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    int input_x = 0;
    int input_y = 0;

    int window_x = 0;
    int window_y = 0;
    
    int output_x = Col;
    int output_y = Row;

    float window_r[4][4];
    float window_g[4][4];
    float window_b[4][4];

    RGBA_t rgba_val;

    for (window_y = 0; window_y < 4; window_y++)
    {
        for (window_x = 0; window_x < 4; window_x++)
        {
            window_r[window_y][window_x] = 0;
            window_g[window_y][window_x] = 0;
            window_b[window_y][window_x] = 0;
        }
    }

    if(output_y < big_height &&  output_x < big_width)
    {
        //Calculate starting index for windows
        float interpolated_x = (float)(output_x / (scale * 1.0));
        float interpolated_y = (float)(output_y / (scale * 1.0));

        int input_block_start_idx_x = (output_x / scale);
        int input_block_start_idx_y = (output_y / scale);

        float dx = interpolated_x - input_block_start_idx_x;
        float dy = interpolated_y - input_block_start_idx_y;

        //We are within a block of the input image, therefore fill windows
        for(window_y = -1; window_y < 3; window_y++)
        {
            for(window_x = -1; window_x < 3; window_x++)
            {
                //Calculate Input Image index
                input_x = input_block_start_idx_x + window_x;
                input_y = input_block_start_idx_y + window_y;

                // Fill window with Nearest Neighbor edge behavior
                if(input_x < 0 || input_x >= width)
                {
                    // Find nearest in-bounds pixel
                    input_x = (input_x < 0) ? 0 : width - 1;
                }
                // Fill window with Nearest Neighbor edge behavior
                if(input_y < 0 || input_y >= height)
                {
                    // Find nearest in-bounds pixel
                    input_y = (input_y < 0) ? 0 : height - 1;
                }
                rgba_val = img_data[input_y * width + input_x];

                window_r[window_y + 1][window_x + 1] = (float)rgba_val.r;    //R
                window_g[window_y + 1][window_x + 1] = (float)rgba_val.g;    //G
                window_b[window_y + 1][window_x + 1] = (float)rgba_val.b;    //B
            }
        }

        rgba_val.r = (unsigned char)bicubicInterpolateDevice_GreyCon(window_r, dy, dx);
        rgba_val.g = (unsigned char)bicubicInterpolateDevice_GreyCon(window_g, dy, dx);
        rgba_val.b = (unsigned char)bicubicInterpolateDevice_GreyCon(window_b, dy, dx);

        big_img_data[output_y * big_width + output_x] = rgba_val;

        grey_big_img_data[output_y * big_width + output_x] = 0.21f * rgba_val.r + 0.71f * rgba_val.g + 0.07f * rgba_val.b;

    }
}




// __device__ float cubicInterpolateDevice_Shared(float p[4], float x)
// {
//     float output = p[1] + 0.5 * x * (p[2] - p[0] + x * (2.0 * p[0] - 5.0 * p[1] + 4.0 * p[2] - p[3] + x * (3.0 * (p[1] - p[2]) + p[3] - p[0])));

//     output = output * ((output <= 255.0) && (output >= 0.0)) + 255 * (output > 255.0) + 0 * (output < 0);
//     return output;
// }

__device__ float bicubicInterpolateDevice_Shared(float p[4][4], float y, float x)
{
    float arr[4];
    float temp;
    float dx_half   = 0.5 * x;

    temp = p[0][1] + dx_half * (p[0][2] - p[0][0] + x * (2.0 * p[0][0] - 5.0 * p[0][1] + 4.0 * p[0][2] - p[0][3] + x * (3.0 * (p[0][1] - p[0][2]) + p[0][3] - p[0][0])));
    arr[0] = temp;

    temp = p[1][1] + dx_half* (p[1][2] - p[1][0] + x * (2.0 * p[1][0] - 5.0 * p[1][1] + 4.0 * p[1][2] - p[1][3] + x * (3.0 * (p[1][1] - p[1][2]) + p[1][3] - p[1][0])));
    arr[1] = temp;

    temp = p[2][1] + dx_half * (p[2][2] - p[2][0] + x * (2.0 * p[2][0] - 5.0 * p[2][1] + 4.0 * p[2][2] - p[2][3] + x * (3.0 * (p[2][1] - p[2][2]) + p[2][3] - p[2][0])));
    arr[2] = temp;

    temp = p[3][1] + dx_half * (p[3][2] - p[3][0] + x * (2.0 * p[3][0] - 5.0 * p[3][1] + 4.0 * p[3][2] - p[3][3] + x * (3.0 * (p[3][1] - p[3][2]) + p[3][3] - p[3][0])));
    arr[3] = temp;

    temp =  arr[1] + 0.5 * y * (arr[2]  - arr[0]  + y * (2.0 * arr[0]  - 5.0 * arr[1]  + 4.0 * arr[2]  - arr[3]  + y * (3.0 * (arr[1]  - arr[2])  + arr[3]  - arr[0])));

    temp = temp * ((temp < 256.0) && (temp > -1.0)) + 255 * (temp > 255.0);//+ 0 * (temp < 0);
    return temp;
}

//Run with block sizes that are multiples of the scale
__global__ void bicubicInterpolation_Shared_Memory_GreyCon_Kernel_RGBA(RGBA_t* big_img_data, unsigned char* grey_big_img_data, RGBA_t* img_data, int big_width, int big_height, int width, int height, int scale)
{

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
 
    int g_input_x = 0;
    int g_input_y = 0;
    
    int g_output_x = Col;
    int g_output_y = Row;

    int tile_input_x = 0;
    int tile_input_y = 0;
    
    int window_x = 0;
    int window_y = 0;

    //Only based on Block Size
    int tile_width = (blockDim.x / scale) + 3;
    int tile_height = (blockDim.y / scale) + 3;
    extern __shared__ RGBA_t s_tile[];

    float window_r[4][4];
    float window_g[4][4];
    float window_b[4][4];

    RGBA_t rgba_val;

    if(threadIdx.x < tile_width && threadIdx.y < tile_height)
    {
        //Calculate Global Input Index
        g_input_x = blockIdx.x * (blockDim.x / scale) + threadIdx.x - 1;
        g_input_y = blockIdx.y * (blockDim.y / scale) + threadIdx.y - 1;

        // Fill window with Nearest Neighbor edge behavior
        if(g_input_x < 0 || g_input_x >= width)
        {
            // Find nearest in-bounds pixel
            g_input_x = (g_input_x < 0) ? 0 : width - 1;
        }
        // Fill window with Nearest Neighbor edge behavior
        if(g_input_y < 0 || g_input_y >= height)
        {
            // Find nearest in-bounds pixel
            g_input_y = (g_input_y < 0) ? 0 : height - 1;
        }

        s_tile[threadIdx.y * tile_width + threadIdx.x] = img_data[g_input_y * width + g_input_x];
    }
    __syncthreads();

    if(g_output_y < big_height && g_output_x < big_width)
    {
        //Calculate starting index for windows (funky stuff to remove shift)
        float interpolated_x = (((float)threadIdx.x + 0.5f) / (float)scale - 0.5f);
        float interpolated_y = (((float)threadIdx.y + 0.5f) / (float)scale - 0.5f);

        //Round down to nearest index
        int interpolated_idx_x = interpolated_x;
        int interpolated_idx_y = interpolated_y;


        ////Calculate starting index for input tile
        //float interpolated_x = (float)((threadIdx.x / (scale * 1.0)) + 1.0);
        //float interpolated_y = (float)((threadIdx.y / (scale * 1.0)) + 1.0);

        //int interpolated_idx_x = (threadIdx.x / scale) + 1;
        //int interpolated_idx_y = (threadIdx.y / scale) + 1;

        float dx = interpolated_x - interpolated_idx_x;
        float dy = interpolated_y - interpolated_idx_y;

        //Fill local window with tiled input data
        for(window_y = -1; window_y < 3; window_y++)
        {
            for(window_x = -1; window_x < 3; window_x++)
            {
                //Calculate Input Image Tile index
                tile_input_x = interpolated_idx_x + window_x + 1;
                tile_input_y = interpolated_idx_y + window_y + 1;

                rgba_val = s_tile[tile_input_y * tile_width + tile_input_x];

                window_r[window_y + 1][window_x + 1] = (float)rgba_val.r;    //R
                window_g[window_y + 1][window_x + 1] = (float)rgba_val.g;    //G
                window_b[window_y + 1][window_x + 1] = (float)rgba_val.b;    //B
            }
        }

        rgba_val.r = (unsigned char)bicubicInterpolateDevice_Shared(window_r, dy, dx);
        rgba_val.g = (unsigned char)bicubicInterpolateDevice_Shared(window_g, dy, dx);
        rgba_val.b = (unsigned char)bicubicInterpolateDevice_Shared(window_b, dy, dx);

        big_img_data[g_output_y * big_width + g_output_x] = rgba_val;

        grey_big_img_data[g_output_y * big_width + g_output_x] = 0.21f * rgba_val.r + 0.71f * rgba_val.g + 0.07f * rgba_val.b;

    }

}

__global__ void nearestNeighbors_GreyCon_Kernel_RGBA(RGBA_t* big_img_data, unsigned char* grey_big_img_data, RGBA_t* img_data, int big_width, int big_height, int width, int height, int scale)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    int small_x = 0;    int small_y = 0;

    RGBA_t rgba_val;

    if (Row < big_height && Col < big_width)
    {
        small_x = Col / scale;
        small_y = Row / scale;

        rgba_val = img_data[small_y * width + small_x];

        big_img_data[Row * big_width + Col] = rgba_val;

        grey_big_img_data[Row * big_width + Col] = (21 * rgba_val.r / 100) + (71 * rgba_val.g /100) + (7  * rgba_val.b /100);
    }
}

#define WINDOW_SIZE     8
#define WINDOW_PIXELS   64
__global__ void Artifact_Shared_Memory_Kernel(float* artifact_map, unsigned char* img_1, unsigned char* img_2, int width, int height)
{
    //int window_size = 8;
    //Window size dictates the size of structures that we can detect. Maybe should look into what effect this has
    //on overall image quality & performance
    // Consider the gaussian option with an 11x11 window

    extern __shared__ float window_img[];

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    float sum1 = 0, sum2 = 0, sum1Sq = 0, sum2Sq = 0, sum12 = 0;
    float img_diff;

    int valid_count = 0;

    //For now, generate a smaller image.

    if (Row < height && Col < width)
    {
        sum1 = img_1[Row * width + Col];    //Using these as temp registers. Just pretend they are called temp1 & temp2
        sum2 = img_2[Row * width + Col];
        window_img[tid_y * WINDOW_SIZE + tid_x + WINDOW_PIXELS] = sum1;
        window_img[tid_y * WINDOW_SIZE + tid_x] = sum2;
        img_diff = (float)abs((sum1 - sum2) / 255.0);
    }

    else
    {
        window_img[tid_y * WINDOW_SIZE + tid_x + WINDOW_PIXELS] = -1;
        window_img[tid_y * WINDOW_SIZE + tid_x] = -1;
    }

    sum1 = 0; sum2 = 0; //reset registers

    __syncthreads();

    for (int i = 0; i < 8; ++i)
    {
        for (int j = 0; j < 8; ++j)
        {
            if ((window_img[i * WINDOW_SIZE + j + WINDOW_PIXELS] >= 0) && (window_img[i * WINDOW_SIZE + j] >= 0))
            {
                sum1 += window_img[i * WINDOW_SIZE + j + WINDOW_PIXELS];
                sum2 += window_img[i * WINDOW_SIZE + j];
                sum1Sq += window_img[i * WINDOW_SIZE + j + WINDOW_PIXELS] * window_img[i * WINDOW_SIZE + j + WINDOW_PIXELS];
                sum2Sq += window_img[i * WINDOW_SIZE + j] * window_img[i * WINDOW_SIZE + j];
                sum12 += window_img[i * WINDOW_SIZE + j + WINDOW_PIXELS] * window_img[i * WINDOW_SIZE + j];
                valid_count++;
            }
        }
    }

    float mu1 = sum1 / valid_count;
    float mu2 = sum2 / valid_count;
    float sigma1Sq = (sum1Sq / valid_count) - (mu1 * mu1);
    float sigma2Sq = (sum2Sq / valid_count) - (mu2 * mu2);
    float sigma12 = (sum12 / valid_count) - (mu1 * mu2);

    // Stabilizing constants
    float C1 = 6.5025; // (K1*L)^2, where K1=0.01 and L=255
    float C2 = 58.5225; // (K2*L)^2, where K2=0.03 and L=255

    float ssim = ((2 * mu1 * mu2 + C1) * (2 * sigma12 + C2)) / ((mu1 * mu1 + mu2 * mu2 + C1) * (sigma1Sq + sigma2Sq + C2));

    artifact_map[Row * width + Col] = ssim * img_diff;
}

__constant__ float d_guas_kernel_seperable[7] = { 0.0366328470,   0.111280762,    0.216745317,    0.270682156,    0.216745317,    0.111280762,    0.0366328470 };

__global__ void horizontalGAUSSianBlurConvolve(float* blur_map, float* input_map, int width, int height, int ksize)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;

    //Shared Memory based on block size and kernel size (always 7 in our case)
    // Tile Width   = Block_Width + KSize - 1
    // Tile Height  = Block_Height
    extern __shared__ float s_tile_h[];

    int tile_width  = blockDim.x + ksize - 1;
    int tile_height = blockDim.y;
    int radius = ksize/2;


    //Fill Input Tile by striding through the input array
    int tile_input_idx_x = tidx;
    int tile_input_idx_y = tidy;
    while(tile_input_idx_x < tile_width)
    {
        int g_input_idx_x = (tile_input_idx_x - radius) + blockIdx.x * blockDim.x;

        if (g_input_idx_x >= 0 && g_input_idx_x < width) 
        {
            s_tile_h[tile_input_idx_y * tile_width + tile_input_idx_x] = input_map[Row * width + g_input_idx_x];
        }
        else
        {
            s_tile_h[tile_input_idx_y * tile_width + tile_input_idx_x] = 0;
        }

        //Stride by blockDim ammount
        tile_input_idx_x += blockDim.x;
    }
    __syncthreads();


    float sum = 0;
    if (Row < height && Col < width)
    {
        //Define Starting points for input tile
        int tile_y = tidy;
        int tile_x = tidx + radius;

        //Horizontal Convolve
        for (int k = -radius; k <= radius; k++) 
        {
            sum += s_tile_h[tile_y * tile_width + (tile_x - k)] * d_guas_kernel_seperable[k + radius];
        }

        //Global Write
        blur_map[Row * width + Col] = sum;
    }
}

__global__ void verticalGAUSSianBlurConvolve(float* blur_map, float* input_map, int width, int height, float threshold, int ksize)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;

    //Shared Memory based on block size and kernel size (always 7 in our case)
    // Tile Width   = Block_Width
    // Tile Height  = Block_Height + KSize - 1
    extern __shared__ float s_tile_v[];

    int tile_width  = blockDim.x;
    int tile_height = blockDim.y + ksize - 1;
    int radius = ksize/2;


    //Fill Input Tile by striding through the input array
    int tile_input_idx_x = tidx;
    int tile_input_idx_y = tidy;
    while(tile_input_idx_y < tile_height)
    {
        int g_input_idx_y = (tile_input_idx_y - radius) + blockIdx.y * blockDim.y;

        if (g_input_idx_y >= 0 && g_input_idx_y < height) 
        {
            s_tile_v[tile_input_idx_y * tile_width + tile_input_idx_x] = input_map[g_input_idx_y * width + Col];
        }
        else
        {
            s_tile_v[tile_input_idx_y * tile_width + tile_input_idx_x] = 0;
        }

        //Stride by blockDim ammount
        tile_input_idx_y += blockDim.y;
    }
    __syncthreads();


    float sum = 0;
    if (Row < height && Col < width)
    {
        //Define Starting points for input tile
        int tile_y = tidy + radius;
        int tile_x = tidx;

        //Horizontal Convolve
        for (int k = -radius; k <= radius; k++) 
        {
            sum += s_tile_v[(tile_y - k) * tile_width + tile_x] * d_guas_kernel_seperable[k + radius];
        }

        //Global Write
        blur_map[Row * width + Col] = (sum > threshold) ? 1.0 : 0.0;//sum;
    }
}


__global__ void GAUSSianBlur_Threshold_Map_Naive_Kernel(float* blur_map, float* input_map, int width, int height, int radius, float sigma, float threshold)
{
    //Generate Normalized GAUSSian Kernal for blurring. This may need to be adjusted so I'll make it flexible.
    //We can eventually hardcode this when we settle on ideal blur.
    int kernel_size = 2 * radius + 1;
    int kernel_center = kernel_size / 2;
    float sum = 0.0;
    float gaussian_kernel[49] = { 0 };

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    float my_PI = 3.1415926535897932384626433832795028841971693993751058209749445923078164062;

    if (Row < height && Col < width)
    {
        for (int y = 0; y < kernel_size; y++)
        {
            for (int x = 0; x < kernel_size; x++)
            {
                double exponent = -((x - kernel_center) * (x - kernel_center) - (y - kernel_center) * (y - kernel_center)) / (2 * sigma * sigma);
                gaussian_kernel[y * kernel_size + x] = exp(exponent) / (2 * my_PI * sigma * sigma);
                sum += gaussian_kernel[y * kernel_size + x];
            }
        }
        //Normalize
        //May not want to do this as edge cases will not utilize entire kernel.
        //Will try for now. It may be the right way to do it. I don't know for sure.
        for (int i = 0; i < kernel_size; i++)
            for (int j = 0; j < kernel_size; j++)
                gaussian_kernel[i * kernel_size + j] /= sum;

        sum = 0.0;

        for (int i = 0; i < kernel_size; i++) {
            for (int j = 0; j < kernel_size; j++) {
                int map_y = Row + i - radius; //
                int map_x = Col + j - radius;

                //If we are within the image
                if (map_x >= 0 && map_x < width && map_y >= 0 && map_y < height) {
                    //printf("map_x %d, map_y %d\n", map_x, map_y);
                    sum += input_map[map_y * width + map_x] * gaussian_kernel[i * kernel_size + j];
                }
            }
        }

        __syncthreads();
        blur_map[Row * width + Col] = (sum > threshold) ? 1.0 : 0.0;
    }
}

__device__ float calculateSSIM_Device(float window1[8][8], float window2[8][8], int window_width, int window_height)
{
    float sum1 = 0, sum2 = 0, sum1Sq = 0, sum2Sq = 0, sum12 = 0;
    int size = window_height * window_width;
    int valid_count = 0;

    for (int i = 0; i < window_height; ++i)
    {
        for (int j = 0; j < window_width; ++j)
        {
            if ((window1[i][j] >= 0) && (window2[i][j] >= 0))
            {
                sum1 += window1[i][j];
                sum2 += window2[i][j];
                sum1Sq += window1[i][j] * window1[i][j];
                sum2Sq += window2[i][j] * window2[i][j];
                sum12 += window1[i][j] * window2[i][j];
                valid_count++;
            }
        }
    }

    float mu1 = sum1 / valid_count;
    float mu2 = sum2 / valid_count;
    float sigma1Sq = (sum1Sq / valid_count) - (mu1 * mu1);
    float sigma2Sq = (sum2Sq / valid_count) - (mu2 * mu2);
    float sigma12 = (sum12 / valid_count) - (mu1 * mu2);

    // Stabilizing constants
    float C1 = 6.5025; // (K1*L)^2, where K1=0.01 and L=255
    float C2 = 58.5225; // (K2*L)^2, where K2=0.03 and L=255

    float ssim = ((2 * mu1 * mu2 + C1) * (2 * sigma12 + C2)) / ((mu1 * mu1 + mu2 * mu2 + C1) * (sigma1Sq + sigma2Sq + C2));
    return ssim;
}

__global__ void Artifact_Grey_Kernel(float* artifact_map, unsigned char* img_1, unsigned char* img_2, int width, int height)
{
    //int window_size = 8;
    //Window size dictates the size of structures that we can detect. Maybe should look into what effect this has
    //on overall image quality & performance
    // Consider the gaussian option with an 11x11 window
    float window_img1[8][8] = { 0 };
    float window_img2[8][8] = { 0 };

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    //For now, generate a smaller image.
    if (Row < height && Col < width)
    {
        for (int i = 0; i < 8; i++)
        {
            for (int j = 0; j < 8; j++)
            {
                if (((Row + i) * width + (Col + j)) < (width * height))
                {
                    window_img1[i][j] = img_1[(Row + i) * width + (Col + j)];
                    window_img2[i][j] = img_2[(Row + i) * width + (Col + j)];
                }
                else
                {
                    window_img1[i][j] = -1;
                    window_img2[i][j] = -1;
                }
            }
        }

        artifact_map[Row * width + Col] = calculateSSIM_Device(window_img1, window_img2, 8, 8) * (float)abs((window_img1[0][0] - window_img2[0][0]) / 255.0);
    }
}