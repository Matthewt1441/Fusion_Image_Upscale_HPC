#include "hip/hip_runtime.h"
// call each kernel implemented in the kernel.cu
// generates timing info
// tests for functional verification

#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <chrono>

#include "util.cu"
#include "serial_code.cu"
#include "kernel.cu"


char file_path[50];
char file_name[50];
char file_address[50];

char game_name[50];

int scale; 

int start_frame;
int end_frame;
int STREAM_COUNT;


int serial_version()
{
    int width;
    int height;
    int current_frame = start_frame;

    int big_width;
    int big_height;
    int big_pixel_count;

    //Host Array Pointers, these should always be unsigned char
    unsigned char*  h_img;                              //Original Small Input Image
    unsigned char*  h_big_img_nn;                       //Upscaled Nearest Neighbor Image
    unsigned char*  h_big_img_nn_grey;                  //Upscaled Greyscale Nearest Neighbor Image
    unsigned char*  h_big_img_bic;                      //Upscaled Bicubic Image
    unsigned char*  h_big_img_bic_grey;                 //Upscaled Greyscale Bicubic Image
    unsigned char*  h_big_img_DIFF_grey;                //Upscaled Greyscale Difference Image
    unsigned char*  h_big_img_SSIM_grey;                //Upscaled Greyscale SSIM Image
    unsigned char*  h_big_img_ARTIFACT_grey;            //Upscaled Greyscale ARTIFACT Image
    unsigned char*  h_big_img_BLURRED_ARTIFACT_grey;    //Upscaled Greyscale BLURRED ARTIFACT Image
    unsigned char*  h_big_img_fused;                    //Upscaled Fused Image
    float*          h_diff_map;                         //Difference Map
    float*          h_ssim_map;                         //SSIM Map
    float*          h_artifact_map;                     //Artifact Map
    float*          h_blurred_artifact_map;             //Blurred Artifact Map

    //Lets start off with timing one image
    try
    {
        //***** Temp *****//
        double processing_time = 0;
 
        //**************** Setup Kernel ****************//
        h_img = (unsigned char*)readPPM(file_address, &width, &height);
        free(h_img);

        //Define big image width and height
        big_width = width * scale; big_height = height * scale;
        big_pixel_count = big_width * big_height;

        //******** Malloc Host Images ********//
        h_big_img_nn                    = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count * 3);
        h_big_img_nn_grey               = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count);
        h_big_img_bic                   = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count * 3);
        h_big_img_bic_grey              = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count);
        h_big_img_ARTIFACT_grey         = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count);
        h_big_img_BLURRED_ARTIFACT_grey = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count);
        h_big_img_fused                 = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count * 3);
        h_big_img_fused                 = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count * 3);
        h_diff_map                      = (float*)malloc(sizeof(float) * big_pixel_count);
        h_ssim_map                      = (float*)malloc(sizeof(float) * big_pixel_count);
        h_artifact_map                  = (float*)malloc(sizeof(float) * big_pixel_count);
        h_blurred_artifact_map          = (float*)malloc(sizeof(float) * big_pixel_count);
        //******** Malloc Host Images ********//

        float execution_time_ms = 0, execution_time_s;
        float current_fps = 0;
        float average_execution_time = 0;
        float average_fps = 0;

        //******************************* Run & Time CODE ********************************//
        printf("SERIAL Code Test\n");
        printf("Scale Factor, %d, Game, %s, Start Frame, %d, End Frame, %d\n", scale, game_name, start_frame, end_frame);
        printf("Input Image Dimensions, %d , %d, Output Image Dimensions, %d, %d\n", width, height, big_width, big_height);

        //Timer
        auto program_start = std::chrono::high_resolution_clock::now();

        while (current_frame <= end_frame)
        {
            //PHASE 0 : Load Input Image
            memset(file_address, 0, sizeof(file_address));

            strcat(file_address, file_path); 
            sprintf(file_name, "%d.ppm", current_frame);
            strcat(file_address, file_name);

            //printf(file_address); printf("\n");

            h_img = (unsigned char*)readPPM(file_address, &width, &height);

            //Timer
            auto start = std::chrono::high_resolution_clock::now();

            //PHASE 1 : IMAGE UPSCALE WITH NEAREST NEIGHBORS & BICUBIC
            nearestNeighbors(h_big_img_nn, big_width, big_height, h_img, width, height, scale);
            bicubicInterpolation(h_big_img_bic, big_width, big_height, h_img, width, height, scale);

            //PHASE 2 : IMAGE GREY SCALE CONVERSION
            RGB2Greyscale(h_big_img_nn_grey, h_big_img_nn, big_width, big_height);
            RGB2Greyscale(h_big_img_bic_grey, h_big_img_bic, big_width, big_height);
            
            //PHASE 3 : ARTIFACT MAP CREATION
            ABS_Difference_Grey(h_diff_map, h_big_img_nn_grey, h_big_img_bic_grey, big_width, big_height);
            SSIM_Grey(h_ssim_map, h_big_img_nn_grey, h_big_img_bic_grey, big_width, big_height);
            MapMul(h_artifact_map, h_diff_map, h_ssim_map, big_width, big_height);

            //PHASE 4 : MAP BLUR
            GaussianBlur_Map(h_blurred_artifact_map, h_artifact_map, big_width, big_height, 3, 1.5);
            MapThreshold(h_blurred_artifact_map, 0.05, big_width, big_height);

            //PHASE 5 : IMAGE FUSION
            Image_Fusion(h_big_img_fused, h_big_img_nn, h_big_img_bic, h_blurred_artifact_map, big_width, big_height);

            auto end = std::chrono::high_resolution_clock::now();
            auto dur = end - start;
            processing_time = std::chrono::duration_cast<std::chrono::microseconds>(dur).count();
            
            execution_time_ms = processing_time / 1000;
            execution_time_s = execution_time_ms / 1000;
            current_fps = 1/(execution_time_s);
            average_execution_time += execution_time_s;
            average_fps = 1/(average_execution_time / (current_frame - start_frame + 1));

            printf("SERIAL CODE: FRAME, %d, TIME, %f, ms, CURRENT FPS, %f, AVERAGE FPS, %f\n", current_frame, execution_time_ms, current_fps, average_fps);
            current_frame++;
        }

        auto program_end = std::chrono::high_resolution_clock::now();
        auto dur = program_end - program_start;
        processing_time = std::chrono::duration_cast<std::chrono::milliseconds>(dur).count();
        processing_time /= 1000; // ms / 1000 -> s

        //************************* CLEAN UP *****************************//
        printf("FINAL SERIAL CODE FPS, %f, Total Execution Time for %d frames, %f, s\n", average_fps, end_frame - start_frame + 1, processing_time);
        writePPM("./MAIN_OUTPUT/SERIAL_OUTPUT/FUSED.ppm", (char*)h_big_img_fused, big_width, big_height);


        //Free Host Memory
        free(h_img);                
        free(h_big_img_nn);         free(h_big_img_bic);   
        free(h_big_img_nn_grey);    free(h_big_img_bic_grey);
        free(h_diff_map);           free(h_ssim_map);
        free(h_artifact_map);       free(h_blurred_artifact_map);

        free(h_big_img_fused);
    }

    catch (const std::exception& e)
    {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    catch (char const* err)
    {
        printf(err); printf("\n");
        return 1;
    }

    return 0;
}

int optimized_version()
{
    int current_frame = start_frame;
    
    int width;
    int height;

    int big_width;
    int big_height;
    int big_pixel_count;

    //Host Array Pointers, these should always be unsigned char
    unsigned char*  h_img;                              //Original Small Input Image
    unsigned char*  h_big_img_fused;                    //Upscaled Fused Image

    //Device Array Pointers
    unsigned char*      d_img;                              //Original Small Input Image
    RGBA_t*             d_RGBA_img;                         //Original Small Input Image w/ 32bit-pixel format
    RGBA_t*             d_big_img_nn;                       //Upscaled Nearest Neighbor Image w/ 32bit-pixel format
    RGBA_t*             d_big_img_bic;                      //Upscaled Bicubic Image w/ 32bit-pixel format
    unsigned char*      d_big_img_nn_grey;                  //Upscaled Greyscale Nearest Neighbor Image
    unsigned char*      d_big_img_bic_grey;                 //Upscaled Greyscale Bicubic Image
    float*              d_big_artifact_map;                 //Upscaled Artifact Map for image fusion
    float*              d_big_blurred_artifact_map;         //Upscaled Blurred Artifact Map for image fusion
    float*              d_big_blurred_artifact_map_inter;   //Upscaled Blurred Artifact Map for image fusion
    RGBA_t*             d_big_rgba_img_fused;               //Upscaled Fused Image w/ 32bit-pixel format
    unsigned char*      d_big_img_fused;                    //Upscaled Fused Image  
    

    //Lets start off with timing one image
    try
    {
        //Check that CUDA-capable GPU is installed
        hipError_t cudaStatus;
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        }


        float processing_time = 0;
 
        //**************** Setup Kernel ****************//
        h_img = (unsigned char*)readPPM(file_address, &width, &height);
        free(h_img);

        //Define big image width and height
        big_width = width * scale; big_height = height * scale;
        big_pixel_count = big_width * big_height;

        //******** Malloc Host Images ********//
        h_big_img_fused                 = (unsigned char*)malloc(sizeof(unsigned char) * big_pixel_count * 3);
        //******** Malloc Host Images ********//


        //******** Malloc Device Images ********//
        //Original Image & RGBA Image
        if (hipMalloc((void**)&d_img, width * height * sizeof(unsigned char) * 3) != hipSuccess)
            fprintf(stderr, "Original Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
        if (hipMalloc((void**)&d_RGBA_img, width * height * sizeof(RGBA_t)) != hipSuccess)
            fprintf(stderr, "RGBA Original Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

        //Upscaled Images
        if (hipMalloc((void**)&d_big_img_nn, big_width * big_height * sizeof(RGBA_t)) != hipSuccess)
            fprintf(stderr, "NN Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
        if (hipMalloc((void**)&d_big_img_bic, big_width * big_height * sizeof(RGBA_t)) != hipSuccess)
            fprintf(stderr, "BIC Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

        //Grey Versions for Upscaled Images
        if (hipMalloc((void**)&d_big_img_nn_grey, big_width * big_height * sizeof(unsigned char)) != hipSuccess)
            fprintf(stderr, "NN Grey Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
        if (hipMalloc((void**)&d_big_img_bic_grey, big_width * big_height * sizeof(unsigned char)) != hipSuccess)
            fprintf(stderr, "BIC Grey Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

        //Maps for Fusion
        if (hipMalloc((void**)&d_big_artifact_map, big_width * big_height * sizeof(float)) != hipSuccess)
            fprintf(stderr, "Artifact Map Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
        if (hipMalloc((void**)&d_big_blurred_artifact_map_inter, big_width * big_height * sizeof(float)) != hipSuccess)
            fprintf(stderr, "Intermediate Blured Artifact Map Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
        if (hipMalloc((void**)&d_big_blurred_artifact_map, big_width * big_height * sizeof(float)) != hipSuccess)
            fprintf(stderr, "Blured Artifact Map Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

        //Final Image and RGBA Image
        if (hipMalloc((void**)&d_big_img_fused, big_width * big_height * sizeof(unsigned char) * 3) != hipSuccess)
            fprintf(stderr, "Fused Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
        if (hipMalloc((void**)&d_big_rgba_img_fused, big_width * big_height * sizeof(RGBA_t)) != hipSuccess)
            fprintf(stderr, "RGBA Fused Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
        //******** Malloc Device Images ********//

        dim3 RGB_Block(256);
        dim3 RGB_Grid(ceil((big_width * big_height) / (float)RGB_Block.x));
        int  rgbToRGBA_Shared_Mem_Size = sizeof(unsigned char)*RGB_Block.x * 3;
        int  rgbaToRGB_Shared_Mem_Size = sizeof(RGBA_t)*RGB_Block.x;

        dim3 NN_Block(16, 16);
        dim3 NN_Grid(((big_width - 1) / NN_Block.x) + 1, ((big_height - 1) / NN_Block.y) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double
 
        dim3 BiCubic_Block(4*scale, 4*scale);
        dim3 BiCubic_Grid(((big_width - 1) / BiCubic_Block.x) + 1, ((big_height - 1) / BiCubic_Block.y) + 1);
        int  BiCubic_Shared_Mem_Size = sizeof(RGBA_t) * ((BiCubic_Block.y / scale) + 3) * ((BiCubic_Block.x / scale) + 3);
        
        dim3 Arti_Block(8, 8);
        dim3 Arti_Grid(((big_width - 1) / Arti_Block.x) + 1, ((big_height - 1) / Arti_Block.y) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double
        int  Arti_Shared_Mem_Size = sizeof(float) * 2 * 8 * 8;

        //Setup Gaussian Blur based on passed in Args
        int GAUSS_Ksize = 7;
        float GAUSS_Sigma = 1.5;
        dim3 h_Gauss_Block(256, 1);
        dim3 h_Gauss_Grid(((big_width - 1) / h_Gauss_Block.x) + 1, ((big_height - 1) / h_Gauss_Block.y) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double
        dim3 v_Gauss_Block(8, 32);
        dim3 v_Gauss_Grid(((big_width - 1) / v_Gauss_Block.x) + 1, ((big_height - 1) / v_Gauss_Block.y) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double

        hipError_t error;
        //**************** Setup Kernel ****************//

        //Variables for timing
        hipEvent_t total_Time_Start, total_Time_End, one_Frame_Start, one_Frame_End, compute_Frame_Start, compute_Frame_End;

        hipEventCreate(&total_Time_Start);         hipEventCreate(&total_Time_End);        
        hipEventCreate(&one_Frame_Start);          hipEventCreate(&one_Frame_End);
        hipEventCreate(&compute_Frame_Start);      hipEventCreate(&compute_Frame_End);

        float execution_time_ms = 0, execution_time_s;
        float current_fps = 0;
        float average_execution_time = 0, average_execution_time_compute = 0;
        float average_fps = 0;

        //******************************* Run & Time CODE ********************************//
        printf("OPTIMIZED CUDA Code Test\n");
        printf("Scale Factor, %d, Game, %s, Start Frame, %d, End Frame, %d\n", scale, game_name, start_frame, end_frame);
        printf("Input Image Dimensions, %d , %d, Output Image Dimensions, %d, %d\n", width, height, big_width, big_height);

        hipEventRecord(total_Time_Start, 0);           //BEGIN TOTAL EXECUTION TIME

        while (current_frame <= end_frame)
        {
            hipEventRecord(one_Frame_Start, 0);        //BEGIN COMPLETE FRAME TIMING

            //PHASE 0 : Load Input Image
            memset(file_address, 0, sizeof(file_address));

            strcat(file_address, file_path); 
            sprintf(file_name, "%d.ppm", current_frame);
            strcat(file_address, file_name);
            //printf(file_address); printf("\n");

            h_img = (unsigned char*)readPPM(file_address, &width, &height);

            //Copy Input Image to Device
            hipMemcpy(d_img, h_img, sizeof(unsigned char) * width * height * 3, hipMemcpyHostToDevice);
            hipDeviceSynchronize();

            //PHASE 1 : Image Pre-Processing
            hipEventRecord(compute_Frame_Start, 0);    //BEGIN COMPUTATION FRAME TIMING
                                                        //Convert original image to RGBA image
            rgbToRGBA_Kernel << < RGB_Grid, RGB_Block, rgbToRGBA_Shared_Mem_Size >> > (d_RGBA_img, d_img, width * height);

            //PHASE 2 : Image Scaling
            nearestNeighbors_GreyCon_Kernel_RGBA                    <<< NN_Grid, NN_Block >>>                                   
                                (d_big_img_nn, d_big_img_nn_grey, d_RGBA_img, big_width, big_height, width, height, scale);
            bicubicInterpolation_Shared_Memory_GreyCon_Kernel_RGBA  <<< BiCubic_Grid, BiCubic_Block, BiCubic_Shared_Mem_Size >>>
                                (d_big_img_bic, d_big_img_bic_grey, d_RGBA_img, big_width, big_height, width, height, scale);

            //PHASE 3 : Image Artifact Detection
            Artifact_Shared_Memory_Kernel                           <<< Arti_Grid, Arti_Block, Arti_Shared_Mem_Size >>>         
                                (d_big_artifact_map, d_big_img_nn_grey, d_big_img_bic_grey, big_width, big_height);


            //PHASE 4 : Artifact Map Post Processing
            horizontalGaussianBlurConvolve  <<< h_Gauss_Grid, h_Gauss_Block, sizeof(float) * (h_Gauss_Block.x + GAUSS_Ksize - 1) * h_Gauss_Block.y >>>
                                (d_big_blurred_artifact_map_inter, d_big_artifact_map, big_width, big_height, GAUSS_Ksize);


            verticalGaussianBlurConvolve    <<< v_Gauss_Grid, v_Gauss_Block, sizeof(float) * (v_Gauss_Block.y + GAUSS_Ksize - 1) * v_Gauss_Block.x >>>
                                (d_big_blurred_artifact_map, d_big_blurred_artifact_map_inter, big_width, big_height, 0.05, GAUSS_Ksize);
            

            //PHASE 5 : Image Fusion
            Image_Fusion_Kernel_RGBA <<< RGB_Grid, RGB_Block >>>            
                                (d_big_rgba_img_fused       , d_big_img_nn, d_big_img_bic   , d_big_blurred_artifact_map, big_width, big_height);


            //PHASE 6 : Image Post Processing -> Convert Into Original Data Type
            rgbaToRGB_Kernel <<< RGB_Grid, RGB_Block, rgbaToRGB_Shared_Mem_Size>>> (d_big_img_fused, d_big_rgba_img_fused, big_width * big_height);
            hipEventRecord(compute_Frame_End, 0);
            hipEventSynchronize(compute_Frame_End);
            
            //Send Device Images to Host
            hipDeviceSynchronize();
            hipMemcpy(h_big_img_fused, d_big_img_fused, sizeof(unsigned char) * big_width * big_height * 3, hipMemcpyDeviceToHost);
            hipEventRecord(one_Frame_End, 0);
            hipEventSynchronize(one_Frame_End);
            
            hipEventElapsedTime(&processing_time, compute_Frame_Start, compute_Frame_End);

            execution_time_ms = processing_time;
            execution_time_s = execution_time_ms / 1000;
            current_fps = 1/(execution_time_s);
            average_execution_time_compute += execution_time_s;
            average_fps = 1/(average_execution_time_compute / (current_frame - start_frame + 1));

            printf("CUDA CODE COMPUTE: FRAME, %d, TIME, %f, ms, CURRENT FPS, %f, AVERAGE FPS, %f\n", current_frame, execution_time_ms, current_fps, average_fps);

            hipEventElapsedTime(&processing_time, one_Frame_Start, one_Frame_End);
            execution_time_ms = processing_time;
            execution_time_s = execution_time_ms / 1000;
            current_fps = 1/(execution_time_s);
            average_execution_time += execution_time_s;
            average_fps = 1/(average_execution_time / (current_frame - start_frame + 1));

            printf("CUDA CODE TOTAL: FRAME, %d, TIME, %f, ms, CURRENT FPS, %f, AVERAGE FPS, %f\n", current_frame, execution_time_ms, current_fps, average_fps);

            current_frame++;
        }
        hipEventRecord(total_Time_End, 0);
        hipEventSynchronize(total_Time_End);
        hipEventElapsedTime(&processing_time, compute_Frame_Start, total_Time_End);
        execution_time_s = processing_time / 1000;

        int     total_frames    = end_frame - start_frame + 1;
        double  compute_fps     = total_frames / average_execution_time_compute;
        double  total_fps       = total_frames / average_execution_time;

        printf("FINAL CUDA OPTIMIZED CODE COMPUTE FPS, %f, Total Execution Time for %d frames, %f, s\n", compute_fps, end_frame - start_frame + 1, execution_time_s);
        printf("FINAL CUDA OPTIMIZED CODE FPS, %f, Total Execution Time for %d frames, %f, s\n", average_fps, end_frame - start_frame + 1, execution_time_s);

        memset(file_address, 0, sizeof(file_address));

        strcat(file_address, "./MAIN_OUTPUT/OPTIMIZED_OUTPUT/FUSED_"); 

        sprintf(file_name, "Scale_%d_Game_%s", scale, game_name);

        strcat(file_address, file_name);
        strcat(file_address, ".ppm");



        writePPM(file_address, (char*)h_big_img_fused, big_width, big_height);

        //************************* CLEAN UP *****************************//
        // 
        //Free Host Memory
        free(h_img);
        free(h_big_img_fused);

        //Free device Memory
        hipFree(d_img);
        hipFree(d_RGBA_img);
        hipFree(d_big_img_nn);
        hipFree(d_big_img_bic);
        hipFree(d_big_img_nn_grey);
        hipFree(d_big_img_bic_grey);
        hipFree(d_big_artifact_map);
        hipFree(d_big_blurred_artifact_map);
        hipFree(d_big_blurred_artifact_map_inter);
        hipFree(d_big_rgba_img_fused);
        hipFree(d_big_img_fused);
    }

    catch (const std::exception& e)
    {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }

    hipDeviceReset();
    return 0;

}

int streamed_version()
{
    int current_frame = start_frame;

    int width;
    int height;

    int big_width;
    int big_height;
    int big_pixel_count;

    //STREAM 1 POINTERS
    //Host Array Pointers, these should always be unsigned char
    unsigned char*      h_img[STREAM_COUNT];                            //Original Small Input Image
    unsigned char*      h_big_img_fused[STREAM_COUNT];                  //Upscaled Fused Image

    //Device Array Pointers
    unsigned char*      d_img[STREAM_COUNT];                            //Original Small Input Image
    RGBA_t*             d_RGBA_img[STREAM_COUNT];                       //Original Small Input Image w/ 32bit-pixel format
    RGBA_t*             d_big_img_nn[STREAM_COUNT];                     //Upscaled Nearest Neighbor Image w/ 32bit-pixel format
    RGBA_t*             d_big_img_bic[STREAM_COUNT];                    //Upscaled Bicubic Image w/ 32bit-pixel format
    unsigned char*      d_big_img_nn_grey[STREAM_COUNT];                //Upscaled Greyscale Nearest Neighbor Image
    unsigned char*      d_big_img_bic_grey[STREAM_COUNT];               //Upscaled Greyscale Bicubic Image
    float*              d_big_artifact_map[STREAM_COUNT];               //Upscaled Artifact Map for image fusion
    float*              d_big_blurred_artifact_map[STREAM_COUNT];       //Upscaled Blurred Artifact Map for image fusion
    float*              d_big_blurred_artifact_map_inter[STREAM_COUNT]; //Upscaled Blurred Artifact Map for image fusion
    RGBA_t*             d_big_rgba_img_fused[STREAM_COUNT];             //Upscaled Fused Image w/ 32bit-pixel format
    unsigned char*      d_big_img_fused[STREAM_COUNT];                  //Upscaled Fused Image  

    hipStream_t stream[STREAM_COUNT];

    //Timer
    auto program_start = std::chrono::high_resolution_clock::now();

    try
    {
        //Check that CUDA-capable GPU is installed
        hipError_t cudaStatus;
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			exit(-1);
        }

        float processing_time = 0;
 
        //**************** Setup Kernel ****************//

        unsigned char* h_img_dim = (unsigned char*)readPPM(file_address, &width, &height);
        free(h_img_dim);

        //Define big image width and height
        big_width = width * scale; big_height = height * scale;
        big_pixel_count = big_width * big_height;
        int pixel_count = width * height;

        for (int s = 0; s < STREAM_COUNT; s++)
        {
            hipStreamCreate(&stream[s]);

            //******** Malloc Host Images ********//
            hipHostAlloc((void**) &h_img[s], sizeof(unsigned char) * pixel_count * 3, hipHostMallocDefault);
            hipHostAlloc((void**) &h_big_img_fused[s], sizeof(unsigned char) * big_pixel_count * 3 , hipHostMallocDefault);
            //******** Malloc Host Images ********//
        
            //******** Malloc Device Images ********//
            //Original Image & RGBA Image
            if (hipMalloc((void**)&d_img[s], width * height * sizeof(unsigned char) * 3) != hipSuccess)
                fprintf(stderr, "Original Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
            if (hipMalloc((void**)&d_RGBA_img[s], width * height * sizeof(RGBA_t)) != hipSuccess)
                fprintf(stderr, "RGBA Original Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

            //Upscaled Images
            if (hipMalloc((void**)&d_big_img_nn[s], big_width * big_height * sizeof(RGBA_t)) != hipSuccess)
                fprintf(stderr, "NN Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
            if (hipMalloc((void**)&d_big_img_bic[s], big_width * big_height * sizeof(RGBA_t)) != hipSuccess)
                fprintf(stderr, "BIC Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

            //Grey Versions for Upscaled Images
            if (hipMalloc((void**)&d_big_img_nn_grey[s], big_width * big_height * sizeof(unsigned char)) != hipSuccess)
                fprintf(stderr, "NN Grey Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
            if (hipMalloc((void**)&d_big_img_bic_grey[s], big_width * big_height * sizeof(unsigned char)) != hipSuccess)
                fprintf(stderr, "BIC Grey Big Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

            //Maps for Fusion
            if (hipMalloc((void**)&d_big_artifact_map[s], big_width * big_height * sizeof(float)) != hipSuccess)
                fprintf(stderr, "Artifact Map Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
            if (hipMalloc((void**)&d_big_blurred_artifact_map_inter[s], big_width * big_height * sizeof(float)) != hipSuccess)
                fprintf(stderr, "Intermediate Blured Artifact Map Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
            if (hipMalloc((void**)&d_big_blurred_artifact_map[s], big_width * big_height * sizeof(float)) != hipSuccess)
                fprintf(stderr, "Blured Artifact Map Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));

            //Final Image and RGBA Image
            if (hipMalloc((void**)&d_big_img_fused[s], big_width * big_height * sizeof(unsigned char) * 3) != hipSuccess)
                fprintf(stderr, "Fused Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
            if (hipMalloc((void**)&d_big_rgba_img_fused[s], big_width * big_height * sizeof(RGBA_t)) != hipSuccess)
                fprintf(stderr, "RGBA Fused Image Failed to Malloc: %s\n", hipGetErrorString(cudaStatus));
        }

        dim3 RGB_Block(256);
        dim3 RGB_Grid(ceil((big_width * big_height) / (float)RGB_Block.x));
        int  rgbToRGBA_Shared_Mem_Size = sizeof(unsigned char)*RGB_Block.x * 3;
        int  rgbaToRGB_Shared_Mem_Size = sizeof(RGBA_t)*RGB_Block.x;

        dim3 NN_Block(16, 16);
        dim3 NN_Grid(((big_width - 1) / NN_Block.x) + 1, ((big_height - 1) / NN_Block.y) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double
 
        dim3 BiCubic_Block(4*scale, 4*scale);
        dim3 BiCubic_Grid(((big_width - 1) / BiCubic_Block.x) + 1, ((big_height - 1) / BiCubic_Block.y) + 1);
        int  BiCubic_Shared_Mem_Size = sizeof(RGBA_t) * ((BiCubic_Block.y / scale) + 3) * ((BiCubic_Block.x / scale) + 3);
        
        dim3 Arti_Block(8, 8);
        dim3 Arti_Grid(((big_width - 1) / Arti_Block.x) + 1, ((big_height - 1) / Arti_Block.y) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double
        int  Arti_Shared_Mem_Size = sizeof(float) * 2 * 8 * 8;

        //Setup Gaussian Blur based on passed in Args
        int GAUSS_Ksize = 7;
        float GAUSS_Sigma = 1.5;
        dim3 h_Gauss_Block(256, 1);
        dim3 h_Gauss_Grid(((big_width - 1) / h_Gauss_Block.x) + 1, ((big_height - 1) / h_Gauss_Block.y) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double
        dim3 v_Gauss_Block(8, 32);
        dim3 v_Gauss_Grid(((big_width - 1) / v_Gauss_Block.x) + 1, ((big_height - 1) / v_Gauss_Block.y) + 1);     //Calculate the number of blocks needed for the dimension. 1.0 * Forces Double
        int  h_Gauss_Mem = sizeof(float) * (h_Gauss_Block.x + GAUSS_Ksize - 1) * h_Gauss_Block.y;
        int  v_Gauss_Mem = sizeof(float) * (v_Gauss_Block.y + GAUSS_Ksize - 1) * v_Gauss_Block.x;

        //**************** Setup Kernel ****************//

        //******************************* Run & Time CODE ********************************//
        printf("STREAMED CUDA Code Test\n");
        printf("Scale Factor, %d, Game, %s, Start Frame, %d, End Frame, %d\n", scale, game_name, start_frame, end_frame);
        printf("Input Image Dimensions, %d , %d, Output Image Dimensions, %d, %d\n", width, height, big_width, big_height);

        auto compute_start = std::chrono::high_resolution_clock::now();

        while (current_frame <= end_frame)
        {
            for (int s = 0; s < STREAM_COUNT; s++)
            {
                //PHASE 0 : Load Input Image
                memset(file_address, 0, sizeof(file_address));

                strcat(file_address, file_path); 
                sprintf(file_name, "%d.ppm", current_frame + s);
                strcat(file_address, file_name);

                h_img[s] = (unsigned char*)readPPM(file_address, &width, &height);

                //Copy Input Image to Device
                hipMemcpyAsync(d_img[s], h_img[s], sizeof(unsigned char) * width * height * 3, hipMemcpyHostToDevice, stream[s]);

                //PHASE 1 : Image Pre-Processing : Convert original image to RGBA image
                rgbToRGBA_Kernel << < RGB_Grid, RGB_Block, rgbToRGBA_Shared_Mem_Size, stream[s] >> > (d_RGBA_img[s], d_img[s], width * height);

                //PHASE 2 : Image Scaling
                nearestNeighbors_GreyCon_Kernel_RGBA <<< NN_Grid, NN_Block, 0, stream[s] >>>                                   
                                    (d_big_img_nn[s], d_big_img_nn_grey[s], d_RGBA_img[s], big_width, big_height, width, height, scale);

                bicubicInterpolation_Shared_Memory_GreyCon_Kernel_RGBA  <<< BiCubic_Grid, BiCubic_Block, BiCubic_Shared_Mem_Size, stream[s] >>>
                                    (d_big_img_bic[s], d_big_img_bic_grey[s], d_RGBA_img[s], big_width, big_height, width, height, scale);

                //PHASE 3 : Image Artifact Detection
                Artifact_Shared_Memory_Kernel<<< Arti_Grid, Arti_Block, Arti_Shared_Mem_Size, stream[s] >>>         
                                    (d_big_artifact_map[s], d_big_img_nn_grey[s], d_big_img_bic_grey[s], big_width, big_height);

                //PHASE 4 : Artifact Map Post Processing
                horizontalGaussianBlurConvolve  <<< h_Gauss_Grid, h_Gauss_Block, h_Gauss_Mem, stream[s] >>>
                                    (d_big_blurred_artifact_map_inter[s], d_big_artifact_map[s], big_width, big_height, GAUSS_Ksize);

                verticalGaussianBlurConvolve    <<< v_Gauss_Grid, v_Gauss_Block, v_Gauss_Mem, stream[s] >>>
                                    (d_big_blurred_artifact_map[s], d_big_blurred_artifact_map_inter[s], big_width, big_height, 0.05, GAUSS_Ksize);

                //PHASE 5 : Image Fusion
                Image_Fusion_Kernel_RGBA <<< RGB_Grid, RGB_Block, 0, stream[s] >>>            
                                    (d_big_rgba_img_fused[s], d_big_img_nn[s], d_big_img_bic[s], d_big_blurred_artifact_map[s], big_width, big_height);

                //PHASE 6 : Image Post Processing -> Convert Into Original Data Type
                rgbaToRGB_Kernel <<< RGB_Grid, RGB_Block, rgbaToRGB_Shared_Mem_Size, stream[s]>>> 
                                    (d_big_img_fused[s], d_big_rgba_img_fused[s], big_width * big_height);

                //Send Device Images to Host
                hipMemcpyAsync(h_big_img_fused[s], d_big_img_fused[s], sizeof(unsigned char) * big_width * big_height * 3, hipMemcpyDeviceToHost, stream[s]);
    
            }
            current_frame += STREAM_COUNT;
        }

        auto    program_end = std::chrono::high_resolution_clock::now();
        
        auto    compute_dur = program_end - compute_start;
        auto    program_dur = program_end - program_start;

        double  compute_time = std::chrono::duration_cast<std::chrono::milliseconds>(compute_dur).count();
        double  program_time = std::chrono::duration_cast<std::chrono::milliseconds>(program_dur).count();

        double  compute_execution_time_s = compute_time / 1000;
        double  program_execution_time_s = program_time / 1000;

        int     total_frames    = current_frame - start_frame;

        double  total_fps_compute   = total_frames / compute_execution_time_s;
        double  total_fps_program   = total_frames / program_execution_time_s;

        printf("FINAL CUDA STREAM CODE, STREAMS, %d COMPUTE FPS, %f, Compute Execution Time for %d frames, %f, s\n", 
                                        STREAM_COUNT, total_fps_compute, total_frames, compute_execution_time_s);
        printf("FINAL CUDA STREAM CODE, STREAMS, %d EFFECTIVE FPS, %f, Total Execution Time for %d frames, %f, s\n", 
                                        STREAM_COUNT, total_fps_program, total_frames, program_execution_time_s);

        for (int s = 0; s < STREAM_COUNT; s++)
        {
            //SAVE FINAL FRAMES
            memset(file_address, 0, sizeof(file_address));

            strcat(file_address, "./MAIN_OUTPUT/STREAM_OUTPUT/FUSED_"); 
            sprintf(file_name, "Streams_%d_Scale_%d_Stream_%d.ppm", STREAM_COUNT, scale, s);
            strcat(file_address, file_name);
            
            writePPM(file_address, (char*)h_big_img_fused[s], big_width, big_height);
            
            //************************* CLEAN UP *****************************//
            hipStreamDestroy(stream[s]);

            //Free Host Memory
            hipHostFree(h_img[s]);                          hipHostFree(h_big_img_fused[s]);

            //Free device Memory
            hipFree(d_img[s]);                              hipFree(d_RGBA_img[s]);
            hipFree(d_big_img_nn[s]);                       hipFree(d_big_img_bic[s]);
            hipFree(d_big_img_nn_grey[s]);                  hipFree(d_big_img_bic_grey[s]);
            hipFree(d_big_artifact_map[s]);                 hipFree(d_big_blurred_artifact_map[s]);
            hipFree(d_big_blurred_artifact_map_inter[s]);
            hipFree(d_big_rgba_img_fused[s]);               hipFree(d_big_img_fused[s]);
        }

        unsigned char* Serial_Img = (unsigned char*)readPPM("./MAIN_OUTPUT/SERIAL_OUTPUT/FUSED.ppm", &width, &height);

        //SAVE FINAL FRAMES
        memset(file_address, 0, sizeof(file_address));

        strcat(file_address, "./MAIN_OUTPUT/STREAM_OUTPUT/FUSED_"); 
            sprintf(file_name, "Streams_%d_Scale_%d_Stream_%d.ppm", STREAM_COUNT, scale, STREAM_COUNT-1);
        strcat(file_address, file_name);

        unsigned char* Streamed_Img = (unsigned char*)readPPM(file_address, &width, &height);
        printf("Hi\n");
        Image_Compare(Serial_Img, Streamed_Img, 3, width, height);
        printf("Hi2\n");


        free(Serial_Img);
        free(Streamed_Img);

    }

    catch (const std::exception& e)
    {
        std::cerr << "Error: " << e.what() << std::endl;
        hipDeviceReset();
        return 1;
    }
    
    catch (const char* err)
    {
        printf("%s\n", err);
        hipDeviceReset();
        return 1;
    }
    hipDeviceReset();
    return 0;
    
}

int main(int argc, char* argv[])
{
    // ARGUMENTS (Example Below)
    // ./SERIAL_CODE_TEST_BENCH_Solution 2 ./NV/image "Fallout New Vegas" 1 10
    // EXECUTABLE, SCALE, FILE_FORMAT, GAME_NAME, START_FRAME, END_FRAME, STREAM_COUNT
    //          0,     1,           2,         3,           4,         5,            6

    scale = atoi(argv[1]); 
            
    strcpy(file_path, argv[2]);     //copy file path into the var 
    strcpy(game_name, argv[3]);     //copy game name into the var

    start_frame = atoi(argv[4]);
    end_frame   = atoi(argv[5]);

    STREAM_COUNT = atoi(argv[6]);
    
    memset(file_address, 0, sizeof(file_address));

    strcat(file_address, file_path); 
    sprintf(file_name, "%d.ppm", start_frame);
    strcat(file_address, file_name);

    printf(file_address); printf("\n");

    serial_version();
    optimized_version();
    streamed_version();


    return 0;
}