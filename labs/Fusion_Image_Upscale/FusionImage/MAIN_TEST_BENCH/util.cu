#pragma once
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>

#include <fstream>
#include <string>

typedef struct __align__(4) { // Or alignas(4) in C++11
    unsigned char r;
    unsigned char g;
    unsigned char b;
    unsigned char a; // Padding to ensure 4-byte alignment
}RGBA_t;

char* readPPM(char* filename, int* width, int* height) 
{
    //std::ifstream file(filename, std::ios::binary);

    std::ifstream file(filename, std::ios::binary); // open the file and throw exception if it doesn't exist
    if (file.fail())
        throw "File failed to open";

    std::string magicNumber;
    int maxColorValue;
    int w = 0;
    int h = 0;

    file >> magicNumber;
    file >> w >> h >> maxColorValue;

    file.get(); // skip the trailing white space

    size_t size = w * h * 3;
    char* pixel_data = new char[size];

    file.read(pixel_data, size);

    *width = w;
    *height = h;

    return pixel_data;
}

char* readPPM(char* pixel_data, char* filename, int* width, int* height) 
{
    //std::ifstream file(filename, std::ios::binary);

    std::ifstream file(filename, std::ios::binary); // open the file and throw exception if it doesn't exist
    if (file.fail())
        throw "File failed to open";

    std::string magicNumber;
    int maxColorValue;
    int w = 0;
    int h = 0;

    file >> magicNumber;
    file >> w >> h >> maxColorValue;

    file.get(); // skip the trailing white space

    size_t size = w * h * 3;

    file.read(pixel_data, size);

    *width = w;
    *height = h;

    return pixel_data;
}

char* readPPMGray(char* filename, int* width, int* height) 
{
    //std::ifstream file(filename, std::ios::binary);

    std::ifstream file(filename, std::ios::binary); // open the file and throw exception if it doesn't exist
    if (file.fail())
        throw "File failed to open";

    std::string magicNumber;
    int maxColorValue;
    int w = 0;
    int h = 0;

    file >> magicNumber;
    file >> w >> h >> maxColorValue;

    file.get(); // skip the trailing white space

    size_t size = w * h;
    char* pixel_data = new char[size];

    file.read(pixel_data, size);

    *width = w;
    *height = h;

    return pixel_data;
}

void writePPM(char* filename, char* img_data, int width, int height)
{
    std::ofstream file(filename, std::ios::binary);
    if (file.fail())
        throw "File failed to open";

    file << "P6" << "\n" << width << " " << height << "\n" << 255 << "\n";

    size_t size = (width) * (height) * 3;

    file.write(img_data, size);
}

void writePPMGrey(char* filename, char* img_data, int width, int height)
{
    std::ofstream file(filename, std::ios::binary);
    if (file.fail())
        throw "File failed to open";

    file << "P5" << "\n" << width << " " << height << "\n" << 255 << "\n";

    size_t size = (width) * (height);

    file.write(img_data, size);
}

// Function to write a Grey PPM image
// Parameters
// Pointer to "true img" data
// Pointer to tested img data
// Number of channesl (GREY) and (RGB)
// Pointers to width and height integers for the image

#define GREY_CHN    1
#define RGB_CHN     3

void Image_Compare(unsigned char* true_img, unsigned char* test_img, int chn_count, int width, int height)
{
    int idx = 0;
    int y;  int x;

    // The largest difference that can occur is 255, there are width * height * channel count of pixels
    int total_pixels = 255 * width * height * chn_count;
    float image_difference = 0;

    // For each pixel in the image
    for( y = 0; y < height; y++) 
    {
        for(x = 0; x < width; x++)
        {
            //Calculate the idx
            idx = y * width + x;

            //for each of the channels
            for (int i = 0; i < chn_count; i++)
            {
                //calculate the error of the image
                image_difference += abs((true_img[idx + i] - test_img[idx + i]));
            }
        }
    }

    printf("Accuracy: %f%\n", 100 - (100 * (image_difference / total_pixels)));
}
